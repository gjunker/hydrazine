#include "hip/hip_runtime.h"
/*!
	\file Memory.cu
	\author Gregory Diamos <gregory.diamos@gatech.edu>
	\date Tuesday May 8, 2009
	\brief The source file for basic Cuda memory management functions
*/

#ifndef MEMORY_CU_INCLUDED
#define MEMORY_CU_INCLUDED

#include <hydrazine/cuda/Memory.h>

namespace hydrazine
{

namespace cuda
{
	
	__global__ void memcpy(void* destination, 
		const void* source, size_t length)
	{
		_memcpy<MemcpyBase>( destination, source, length );
	}
	
}

}

#endif

